#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#include "include/train_raymarching.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")

inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}


inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}


inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabs(y), fabs(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}


inline __device__ int mip_from_dt(const float dt, const float grid_size, const float max_cascade) {
    const float mx = dt * grid_size * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}


// rays_o/d: [N, 3]
// grid: [CHHH / 8]
// xyzs, dirs, deltas: [M, 3], [M, 3], [M, 2]
// dirs: [M, 3]
// rays: [N, 3], idx, offset, num_steps
template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,  
    const uint8_t * __restrict__ grid,
    const float scale,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t num_rays, const uint32_t num_cascades, 
    const uint32_t grid_size, const uint32_t max_points,
    const scalar_t* __restrict__ nears, 
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * deltas,
    int * rays, 
    int * counter
) {
    // do 2 passes, first to estimate num of steps
    // take small steps if it seems occupied
    // second to march rays and write points

    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= num_rays) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    // ray marching
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)grid_size;
    const float H3 = grid_size * grid_size * grid_size;
    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];
    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (num_cascades - 1)) / grid_size;
    
    float t0 = near;
    
    // manually clamp TODO: could add noise here
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max)

    // first pass: estimation of num_steps
    float t = t0;
    uint32_t num_steps = 0;

    while (t < far && num_steps < max_steps) {
        // current point
        const float x = clamp(ox + t * dx, -scale, scale);
        const float y = clamp(oy + t * dy, -scale, scale);
        const float z = clamp(oz + t * dz, -scale, scale);
        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level, mipmaps are at different resolutions basically
        const int level = max(mip_from_pos(x, y, z, num_cascades), mip_from_dt(dt, grid_size, num_cascades)); // range in [0, num_cascades - 1]
        const float mip_bound = fminf(scalbnf(1.0f, level), scale);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step
        if (occ) {
            num_steps++;
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }

    // second pass: really locate and write points & dirs
    uint32_t point_index = atomicAdd(counter, num_steps);
    uint32_t ray_index = atomicAdd(counter + 1, 1);

    // write rays
    rays[ray_index * 3] = n;
    rays[ray_index * 3 + 1] = point_index;
    rays[ray_index * 3 + 2] = num_steps;

    if (num_steps == 0) return;
    if (point_index + num_steps > M) return;

    xyzs += point_index * 3;
    dirs += point_index * 3;
    deltas += point_index * 2;
    t = t0;
    uint32_t step = 0;
    float last_t = t;

    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -scale, scale);
        const float y = clamp(oy + t * dy, -scale, scale);
        const float z = clamp(oz + t * dz, -scale, scale);
        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, num_cascades), mip_from_dt(dt, grid_size, num_cascades)); // range in [0, num_cascades - 1]
        const float mip_bound = fminf(scalbnf(1.0f, level), scale);
        const float mip_rbound = 1 / mip_bound;

        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));

        // query grid
        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max); 
            } while (t < tt);
        }
    }
}

void train_raymarching(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor grid, const float scale, const float dt_gamma, const uint32_t max_steps, const uint32_t num_rays, const uint32_t num_cascades, const uint32_t grid_size, const uint32_t max_points, const at::Tensor nears, const at::Tensor fars, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas, at::Tensor rays, at::Tensor counter) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), grid.data_ptr<uint8_t>(), scale, dt_gamma, max_steps, num_rays, num_cascades, grid_size, max_points, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>());
    }));
}
