#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#include "include/train_composite_rays.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")

inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_train_composite_rays_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const uint32_t max_points, const uint32_t num_rays, 
    const float T_thresh, 
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= num_rays) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > max_points) {
        weights_sum[index] = 0;
        depth[index] = 0;
        image[index * 3] = 0;
        image[index * 3 + 1] = 0;
        image[index * 3 + 2] = 0;
        return;
    }
    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;

    // accumulate 
    uint32_t step = 0;
    scalar_t T = 1.0f;
    scalar_t r = 0, g = 0, b = 0, ws = 0, t = 0, d = 0;
    while (step < num_steps) {
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        t += deltas[1]; // real delta
        d += weight * t;
        ws += weight;
        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;

        step++;
    }
    // write
    weights_sum[index] = ws; // weights_sum
    depth[index] = d;
    image[index * 3] = r;
    image[index * 3 + 1] = g;
    image[index * 3 + 2] = b;
}


void train_composite_rays_forward(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const uint32_t max_points, const uint32_t num_rays, const float T_thresh, at::Tensor weights_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), max_points, num_rays, T_thresh, weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights_sum: [N,]
// grad: [N, 3]
// sigmas: [M]
// rgbs: [M, 3]
// deltas: [M, 2]
// rays: [N, 3], idx, offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_train_composite_rays_backward(
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ deltas,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ image,
    const uint32_t max_points, const uint32_t num_rays, const float T_thresh,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= num_rays) return;

    // locate 
    uint32_t index = rays[n * 3];
    uint32_t offset = rays[n * 3 + 1];
    uint32_t num_steps = rays[n * 3 + 2];

    if (num_steps == 0 || offset + num_steps > max_points) return;

    grad_weights_sum += index;
    grad_image += index * 3;
    weights_sum += index;
    image += index * 3;
    sigmas += offset;
    rgbs += offset * 3;
    deltas += offset * 2;
    grad_sigmas += offset;
    grad_rgbs += offset * 3;

    // accumulate 
    uint32_t step = 0;
    
    scalar_t T = 1.0f;
    const scalar_t r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0];
    scalar_t r = 0, g = 0, b = 0, ws = 0;

    while (step < num_steps) {
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);
        const scalar_t weight = alpha * T;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        ws += weight;
        T *= 1.0f - alpha;
        
        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;

        // write grad_sigmas
        grad_sigmas[0] = deltas[0] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
            grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            grad_weights_sum[0] * (1 - ws_final)
        );

        // minimal remained transmittence
        if (T < T_thresh) break;
        
        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        grad_sigmas++;
        grad_rgbs += 3;
        step++;
    }
}

void train_composite_rays_backward(const at::Tensor grad_weights_sum, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor deltas, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor image, const uint32_t max_points, const uint32_t num_rays, const float T_thresh, at::Tensor grad_sigmas, at::Tensor grad_rgbs) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights_sum.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), max_points, num_rays, T_thresh, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>());
    }));
}