#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#include "include/march_rays.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")

inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}


template <typename scalar_t>
__global__ void kernel_march_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const int* __restrict__ rays_alive, 
    const scalar_t* __restrict__ rays_t, 
    const scalar_t* __restrict__ rays_o, 
    const scalar_t* __restrict__ rays_d, 
    const float scale,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t num_cascades, const uint32_t grid_size,
    const uint8_t * __restrict__ grid,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t* xyzs, scalar_t* dirs, scalar_t* deltas
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;
    const int index = rays_alive[n]; // ray id
    const float noise = noises[n];
    
    // locate
    rays_o += index * 3;
    rays_d += index * 3;
    xyzs += n * n_step * 3;
    dirs += n * n_step * 3;
    deltas += n * n_step * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)grid_size;
    const float H3 = grid_size * grid_size * grid_size;
    float t = rays_t[index]; // current ray's t
    const float near = nears[index], far = fars[index];
    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * (1 << (num_cascades - 1)) / grid_size;

    // march for n_step steps, record points
    uint32_t step = 0;

    // introduce some randomness
    t += clamp(t * dt_gamma, dt_min, dt_max) * noise;
    float last_t = t;

    while (t < far && step < n_step) {
        // current point
        const float x = clamp(ox + t * dx, -scale, scale);
        const float y = clamp(oy + t * dy, -scale, scale);
        const float z = clamp(oz + t * dz, -scale, scale);
        const float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, num_cascades), mip_from_dt(dt, grid_size, num_cascades)); // range in [0, C - 1]
        const float mip_bound = fminf(scalbnf(1, level), scale);
        const float mip_rbound = 1 / mip_bound;
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (x * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const int ny = clamp(0.5 * (y * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const int nz = clamp(0.5 * (z * mip_rbound + 1) * H, 0.0f, (float)(grid_size - 1));
        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = x;
            xyzs[1] = y;
            xyzs[2] = z;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            // calc dt
            t += dt;
            deltas[0] = dt;
            deltas[1] = t - last_t; // used to calc depth
            last_t = t;
            // step
            xyzs += 3;
            dirs += 3;
            deltas += 2;
            step++;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - x) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - y) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - z) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                t += clamp(t * dt_gamma, dt_min, dt_max);
            } while (t < tt);
        }
    }
}

void march_rays(const uint32_t n_alive, const uint32_t n_step, const at::Tensor rays_alive, const at::Tensor rays_t, const at::Tensor rays_o, const at::Tensor rays_d, const float scale, const float dt_gamma, const uint32_t max_steps, const uint32_t num_cascades, const uint32_t grid_size, const at::Tensor grid, const at::Tensor near, const at::Tensor far, at::Tensor xyzs, at::Tensor dirs, at::Tensor deltas) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays", ([&] {
        kernel_march_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), scale, dt_gamma, max_steps, num_cascades, grid_size, grid.data_ptr<uint8_t>(), near.data_ptr<scalar_t>(), far.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>());
    }));
}