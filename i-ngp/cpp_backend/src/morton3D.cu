#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>


inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}


inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}


inline __host__ __device__ uint32_t __invert_morton3D(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}


// coords: int32, [N, 3]
// indices: int32, [N]
__global__ void kernel_morton3D(
    const int * __restrict__ coords,
    const uint32_t N,
    int * indices
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;
    indices[n] = __morton3D(coords[0], coords[1], coords[2]);
}


void morton3D(const at::Tensor coords, const uint32_t N, at::Tensor indices) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(coords.data_ptr<int>(), N, indices.data_ptr<int>());
}


// indices: int32, [N]
// coords: int32, [N, 3]
__global__ void kernel_invert_morton3D(
    const int * __restrict__ indices,
    const uint32_t N,
    int * coords
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;

    const int ind = indices[n];

    coords[0] = __invert_morton3D(ind >> 0);
    coords[1] = __invert_morton3D(ind >> 1);
    coords[2] = __invert_morton3D(ind >> 2);
}


void invert_morton3D(const at::Tensor indices, const uint32_t N, at::Tensor coords) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_invert_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(indices.data_ptr<int>(), N, coords.data_ptr<int>());
}


// grid: float, [C, H, H, H]
// N: int, C * H * H * H / 8
// density_thresh: float
// bitfield: uint8, [N]
template <typename scalar_t>
__global__ void kernel_packbits(
    const scalar_t * __restrict__ grid,
    const uint32_t N,
    const float density_thresh,
    uint8_t * bitfield
) {
    // parallel per byte
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grid += n * 8;

    uint8_t bits = 0;

    #pragma unroll
    for (uint8_t i = 0; i < 8; i++) {
        bits |= (grid[i] > density_thresh) ? ((uint8_t)1 << i) : 0;
    }

    bitfield[n] = bits;
}


void packbits(const at::Tensor grid, const uint32_t N, const float density_thresh, at::Tensor bitfield) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grid.scalar_type(), "packbits", ([&] {
        kernel_packbits<<<div_round_up(N, N_THREAD), N_THREAD>>>(grid.data_ptr<scalar_t>(), N, density_thresh, bitfield.data_ptr<uint8_t>());
    }));
}