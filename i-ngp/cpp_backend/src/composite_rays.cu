#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}


template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ deltas, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;
    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    deltas += n * n_step * 2;
    
    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;
    scalar_t t = rays_t[0]; // current ray's t
    scalar_t weight_sum = weights_sum[0];
    scalar_t d = depth[0];
    scalar_t r = image[0];
    scalar_t g = image[1];
    scalar_t b = image[2];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if delta == 0
        if (deltas[0] == 0) break;
        const scalar_t alpha = 1.0f - __expf(- sigmas[0] * deltas[0]);

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const scalar_t T = 1 - weight_sum;
        const scalar_t weight = alpha * T;
        weight_sum += weight;

        t += deltas[1]; // real delta
        d += weight * t;
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        deltas += 2;
        step++;
    }

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }
    
    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
}


void composite_rays(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor deltas, at::Tensor weights, at::Tensor depth, at::Tensor image) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), deltas.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}